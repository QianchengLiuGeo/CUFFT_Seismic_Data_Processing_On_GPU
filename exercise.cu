#include<stdio.h>
#include<stdlib.h>

_global_ void testkernel(int *d_A,size_t size)
{
	int dx=blockDim.x*blockIdx.x+threadIdx.x;
	int dy=blockDim.y*blockIdx.y+threadIdx.y;
	
	if(blockIdx.x==0 && blockIdx.y==0)
		d_A[dx*size+dy]+=1;
	if(blockIdx.x==0 && blockIdx.y==1)
		d_A[dx*size+dy]+=2;
	if(blockIdx.x==1 && blockIdx.y==0)
		d_A[dx*size+dy]+=3;
	if(blockIdx.x==1 && blockIdx.y==1)
		d_A[dx*size+dy]+=4;
}

int main(int argc,char** argv)

{

	int h_A[8][8]={{1,1,1,1,2,2,2,2},
			{1,1,1,1,2,2,2,2},
			{1,1,1,1,2,2,2,2},
			{1,1,1,1,2,2,2,2},
			{3,3,3,3,4,4,4,4},
			{3,3,3,3,4,4,4,4},
			{3,3,3,3,4,4,4,4},
			{3,3,3,3,4,4,4,4}};

	int *d_A,*h_B;
	size_t size=8*8*sizeof(int);
	size_t rsize=8;
	dim3 dimgrid(2,2);
	dim3 dimblock(4,4);
	h_B=(int*)malloc(size);
	
	hipMalloc((void**)&d_A,size);
	hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);

	testkernel<<<dimgrid,dimblock>>>(d_A,rsize);

	hipMemcpy(h_B,d_A,size,hipMemcpyDeviceToHost);

	for(int i=0;i<8;i++){
		for(int j=0;j<8;j++){
			printf("%2d",h_B[i*rsize+j]);	
		}printf("\n");
	}
	hipFree(d_A);
	free(h_B);
}
