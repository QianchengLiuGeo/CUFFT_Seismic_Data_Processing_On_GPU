#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
#include<hip/hip_runtime.h>

#include"pfafft.c"//即使在无法调用su情况下，保证npfar有效
//#include <cutil_inline.h>
//#include <cutil.h>
int main()
{
	int nx,nt,nw,i,ix,it;//nx:the number of traces	nt:the number of time samples	
	int NX,BATCH,NX_t;
	float  **a_input;
	float  *input;
	float *amp;//testing
	hipfftHandle plan;
	hipfftComplex *data;
	clock_t t_z,t_f;
	nx=5300;
	nt=5001;
	NX=npfar(nt);//npfar,from seismic unix
	BATCH=nx;
	nw=NX/2+1;
	NX_t=(NX/2+1)*2;//NX_t makes sure that the array can meet the principle of "in-place"
	input=(float*)calloc(NX_t*BATCH,sizeof(float));
	amp=(float*)calloc(nw,sizeof(float));//testing
	a_input=(float**)calloc(nt,sizeof(float*));
	for(it=0;it<nt;it++){
		a_input[it]=(float*)calloc(nx,sizeof(float));
	}

	FILE *fp;
	fp=fopen("rec_u_3200.bin","rb");
	for(it=0;it<nt;it++){
		for(ix=0;ix<nx;ix++){
			fread(&a_input[it][ix],sizeof(float),1,fp);
		}
	}
	fclose(fp);

	for(ix=0;ix<BATCH;ix++){
		for(it=0;it<nt;it++){
			input[ix*NX_t+it]=a_input[it][ix];
		}
	}
	printf("re_transpose_done !!!\n");

	hipMalloc((void**)&data, sizeof(hipfftComplex)*(NX/2+1)*BATCH);
	hipMemcpy(data,input,NX_t*BATCH*sizeof(float),hipMemcpyHostToDevice);

	t_z=clock();	
	hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH);

	hipfftExecR2C(plan, (hipfftReal*)data, data);
	t_f=clock();
	hipfftDestroy(plan);

	printf("\nCalculating time:%ld (cycles) \n\n",t_f-t_z);
	printf("\nCalculating time:%f (ms) \n\n",(double)(t_f-t_z)/CLOCKS_PER_SEC);

	hipMemcpy(input,data,nx*nt*sizeof(float),hipMemcpyDeviceToHost);
	fp=fopen("bofore_cufft.bin","wb");//testing
	for(it=0;it<nt;it++){
		for(ix=0;ix<1;ix++){
			fwrite(&a_input[it][ix],sizeof(float),1,fp);
		}
	}
	fclose(fp);
	fp=fopen("after_cufft.bin","wb");//testing
	for(it=0;it<nt;it++){
		fwrite(&input[it],sizeof(float),1,fp);
	}
	fclose(fp);	
	hipFree(data);
	for(i=0;i<nt/2;i++){
		amp[i]=sqrt(input[499*NX_t+2*i]*input[499*NX_t+2*i]+input[499*NX_t+2*i+1]*input[499*NX_t+2*i+1]);
	}
	fp=fopen("amp.bin","wb");//testing
	fwrite(amp,sizeof(float),nt/2,fp);
	fclose(fp);
	return 0;

}
