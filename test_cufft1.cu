#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>

//#include <cutil_inline.h>
//#include <cutil.h>
int main()
{
	int nx,nt,i,ix,it;
	int NX,BATCH;
	float  **a_input;
	float  *input;
	float *amp;
	hipfftHandle plan;
	hipfftComplex *data;
	time_t t_z,t_f;
	nx=5300;
	nt=12001;
	a_input=(float**)calloc(nt,sizeof(float*));
	for(it=0;it<nt;it++){
		a_input[it]=(float*)calloc(nx,sizeof(float));
	}
	input=(float*)calloc(nx*nt,sizeof(float));
	amp=(float*)calloc(nt/2,sizeof(float));
	FILE *fp;
	fp=fopen("rec_u_3200.bin","rb");
	for(it=0;it<nt;it++){
		for(ix=0;ix<nx;ix++){
			fread(&a_input[it][ix],sizeof(float),1,fp);
		}
	}
	fclose(fp);

	for(ix=0;ix<nx;ix++){
		for(it=0;it<nt;it++){
			input[ix*nt+it]=a_input[it][ix];
		}
	}
	printf("re_transpose_done !!!\n");
	NX=nt;
	BATCH=10;
	hipMalloc((void**)&data, sizeof(hipfftComplex)*(NX/2+1)*BATCH);
	hipMemcpy(data,input,NX*BATCH*sizeof(float),hipMemcpyHostToDevice);

	t_z=time(NULL);	
	hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH);
	hipfftExecR2C(plan, (hipfftReal*)data, data);
	hipfftDestroy(plan);
	t_f=time(NULL);
	printf("\nCalculating time:%f (s) \n\n",t_f-t_z);
	hipMemcpy(input,data,nx*nt*sizeof(float),hipMemcpyDeviceToHost);
	fp=fopen("bofore_cufft.bin","wb");
	for(it=0;it<nt;it++){
		for(ix=0;ix<1;ix++){
			fwrite(&a_input[it][ix],sizeof(float),1,fp);
		}
	}
	fclose(fp);
	fp=fopen("after_cufft.bin","wb");
	for(it=0;it<nt;it++){
		fwrite(&input[it],sizeof(float),1,fp);
	}
	fclose(fp);	
	hipFree(data);
	for(i=0;i<nt/2;i++){
		amp[i]=sqrt(input[nt+2*i]*input[nt+2*i]+input[nt+2*i+1]*input[nt+2*i+1]);
	}
	fp=fopen("amp.bin","wb");
	fwrite(amp,sizeof(float),nt/2,fp);
	fclose(fp);
	return 0;

}
